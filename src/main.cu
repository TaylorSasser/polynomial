#include <polynomial/polynomial.hpp>
#include <iostream>
#include <random>
#include <iostream>
#include <charconv>
#include <cstdio>
#include <string>
#include <algorithm>
#include <chrono>


#define PROGRAM_EXIT(...) do{ std::fprintf(stderr,__VA_ARGS__); exit(1); } while(0)

template<class T>
auto read_cli_argument(std::string str, char const* error_message)
{
    if constexpr (std::is_arithmetic_v<T>)
    {
        T result{};
        auto [ptr, ec] = std::from_chars(str.data(), str.data() + str.size(), result);
        if (ec != std::errc())
            PROGRAM_EXIT(error_message, str.data());
        return result;
    }
    static_assert("Invalid CLI argument type");
}


int main(int argc, char* argv[])
{
    /*
    if (argc != 3)
        PROGRAM_EXIT("Error: Bad command line parameters\nUsage: ./polynomial <num> <deg>\nEx ./polynomial 10000 50");
    auto const len  = read_cli_argument<std::size_t>(argv[1], "Unable to convert \"%s\" to std::size_t\n");
    auto const deg = read_cli_argument<std::size_t>(argv[2], "Unable to convert \"%s\" to std::size_t\n");
    */
    std::size_t len = (1 << 26);
    std::size_t deg = 10000;

    deg += 1;

    float* host_values = new float[len];
    float* host_coeffs = new float[deg];

    std::fill(host_values + 0, host_values + len, 1);
    std::fill(host_coeffs + 0, host_coeffs + deg, 1);


    float* dev_values = nullptr;
    float* dev_coeffs = nullptr;

    if (auto err = hipMalloc(&dev_values, sizeof(float) * len))
        PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));

    if (auto err = hipMalloc(&dev_coeffs, sizeof(float) * deg))
        PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));

    std::size_t block_dimension = 512;
    std::size_t grid_dimension = (len + block_dimension - 1) / block_dimension;
    std::cout << "Dimension: " << block_dimension << '\n';
    std::cout << "Grid Size: " << grid_dimension << '\n';

    auto beg = std::chrono::system_clock::now();

    if (auto err = hipMemcpy(dev_coeffs, host_coeffs, sizeof(float) * deg, hipMemcpyHostToDevice))
        PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));

    if (auto err = hipMemcpy(dev_values, host_values, sizeof(float) * len, hipMemcpyHostToDevice))
        PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));

    polynomial_expansion<<<grid_dimension,block_dimension>>>(dev_values, dev_coeffs, deg, len);

    if (auto err = hipMemcpy(host_values, dev_values, sizeof(float) * len, hipMemcpyDeviceToHost))
        PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));



    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> total_time = (end - beg);
    double giga_flops = static_cast<double>(2 * deg * len) / total_time.count() / 1e9;

    std::cout << "Seconds: " << total_time.count() << '\n';
    std::cout << "GFlops / Second: " << giga_flops << '\n';

    return 0;
}
