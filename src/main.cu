#include "hip/hip_runtime.h"
#include <polynomial/polynomial.hpp>
#include <iostream>
#include <random>
#include <iostream>
#include <charconv>
#include <cstdio>
#include <string>
#include <algorithm>
#include <chrono>
#include <iomanip>



constexpr static std::size_t const streams{8};

#define PROGRAM_EXIT(...) do{ std::fprintf(stderr,__VA_ARGS__); exit(1); } while(0)

template<class T>
auto read_cli_argument(std::string str, char const* error_message)
{
    if constexpr (std::is_arithmetic_v<T>)
    {
        T result{};
        auto [ptr, ec] = std::from_chars(str.data(), str.data() + str.size(), result);
        if (ec != std::errc())
            PROGRAM_EXIT(error_message, str.data());
        return result;
    }
    static_assert("Invalid CLI argument type");
}


int main(int argc, char* argv[])
{
    if (argc != 3)
        PROGRAM_EXIT("Error: Bad command line parameters\nUsage: ./polynomial <num> <deg>\nEx ./polynomial 10000 50");
    auto const len = (read_cli_argument<std::size_t>(argv[1], "Unable to convert \"%s\" to std::size_t\n") / streams) * streams;
    auto const deg = read_cli_argument<std::size_t>(argv[2], "Unable to convert \"%s\" to std::size_t\n") + 1;

    std::size_t stream_data_len = len / streams;
    std::size_t block_dimension = 512;
    std::size_t grid_dimension = (len + block_dimension - 1) / block_dimension;

    float* host_values = nullptr;
    float* host_coeffs = nullptr;

    if (auto err = hipHostMalloc(&host_values, sizeof(float) * len); err != hipSuccess)
        PROGRAM_EXIT("Error: %s %s\n", hipGetErrorName(err), hipGetErrorString(err));

    if (auto err = hipHostMalloc(&host_coeffs, sizeof(float) * deg); err != hipSuccess)
        PROGRAM_EXIT("Error: %s %s\n", hipGetErrorName(err), hipGetErrorString(err));

    std::fill(host_values + 0, host_values + len, 1);
    std::fill(host_coeffs + 0, host_coeffs + deg, 1);

    float* dev_values = nullptr;
    float* dev_coeffs = nullptr;

    hipStream_t coeffs_stream;
    hipStream_t values_stream[streams];
    hipEvent_t beg_poly[streams];
    hipEvent_t end_poly[streams];

    std::cout << "Streams: " << streams << '\n';
    std::cout << "Dimension: " << block_dimension << '\n';
    std::cout << "Grid Size: " << grid_dimension << '\n';


    if (auto err = hipMalloc(&dev_values, sizeof(float) * len))
        PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));
    if (auto err = hipMalloc(&dev_coeffs, sizeof(float) * deg))
        PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));
    if (auto err = hipStreamCreateWithFlags(&coeffs_stream, hipStreamNonBlocking); err != hipSuccess)
        PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));



    for (std::size_t i = 0; i < streams; i++)
    {
        if (auto err = hipStreamCreateWithFlags(values_stream + i, hipStreamNonBlocking); err != hipSuccess)
            PROGRAM_EXIT("Error: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));
        hipEventCreate(beg_poly + i);
        hipEventCreate(end_poly + i);
    }

    auto beg = std::chrono::high_resolution_clock::now();

    if (auto err = hipMemcpyAsync(dev_coeffs, host_coeffs, deg * sizeof(float), hipMemcpyHostToDevice, coeffs_stream); err != hipSuccess)
        PROGRAM_EXIT("Coeffs Stream: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));

    for (std::size_t i = 0; i < streams; i++)
    {
        std::size_t offset = i * stream_data_len;
        if (auto err = hipMemcpyAsync(dev_values + offset, host_values + offset, stream_data_len * sizeof(float), hipMemcpyHostToDevice, values_stream[i]); err != hipSuccess)
            PROGRAM_EXIT("Values Stream: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));

    }

    //We need to wait before the coeff's are done first.
    hipStreamSynchronize(coeffs_stream);

    for (std::size_t i = 0; i < streams; i++)
    {
        std::size_t offset = i * stream_data_len;
        hipEventRecord(beg_poly[i], values_stream[i]);
        polynomial_expansion<<<grid_dimension / streams, block_dimension, 0, values_stream[i]>>>(dev_values + offset, dev_coeffs, deg, stream_data_len);
        hipEventRecord(end_poly[i], values_stream[i]);
    }

    for (std::size_t i = 0; i < streams; i++)
    {
        std::size_t offset = i * stream_data_len;
        if (auto err = hipMemcpyAsync(host_values + offset, dev_values + offset, stream_data_len * sizeof(float), hipMemcpyDeviceToHost, values_stream[i]); err != hipSuccess)
            PROGRAM_EXIT("Values Stream: %s %s\n ", hipGetErrorName(err), hipGetErrorString(err));
    }

    for (std::size_t i = 0; i < streams; i++)
    {
        hipStreamSynchronize(values_stream[i]);
    }

    auto end = std::chrono::high_resolution_clock::now();

    for (std::size_t i = 0; i < len; i++)
    {
        if (fabs(host_values[i] - static_cast<float>(deg)) > 0.01)
            std::cout << "host_values[" << i << "] should be " << deg << " not " << host_values[i] << '\n';
    }

    std::cout << std::setprecision(16);
    float total_time = 0;
    hipEventElapsedTime(&total_time, beg_poly[streams - 1], end_poly[streams - 1]);
    total_time /= 1e3;

    double giga_flops = static_cast<double>(2 * (deg + 1) * len) / total_time / 1e9;


    std::cout << "Application Seconds: " << std::chrono::duration_cast<std::chrono::seconds>(end - beg).count() << '\n';
    std::cout << "Kernel Seconds: " << total_time << '\n';
    std::cout << "GFlops / Second: " << giga_flops << '\n';

    return 0;
}
